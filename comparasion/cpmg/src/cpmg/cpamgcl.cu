#include <amgcl/amg.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/backend/cuda.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/relaxation/cusparse_ilu0.hpp>  // Slower than spai0
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/cg.hpp>

#include <hip/hip_runtime.h>

#include <pybind11/numpy.h>

#include "cpmg/assembler.h"
#include "cpmg/cpamgcl.h"
#include "cpmg/cpmg.h"


namespace cpmg::cpamgcl
{

namespace
{

using Backend = amgcl::backend::cuda<Float>;

using Preconditioner = amgcl::amg<
        Backend,
        amgcl::coarsening::smoothed_aggregation,
        amgcl::relaxation::spai0
>;

using IterativeSolver = amgcl::solver::cg<Backend>;

using Solver = amgcl::make_solver<Preconditioner, IterativeSolver>;


pybind11::tuple
amgcl_solve(
        const pybind11::array_t<bool> & boundaryMask,
        const pybind11::array_t<Float> & laplacian,
        const pybind11::array_t<Float> & boundaryValue,
        const pybind11::array_t<Float> & initialGuess,
        Float relativeTolerance)
{
    std::vector<int> col, ptr;
    std::vector<Float> val, rhs;

    int n2 = assemble2DPoissonProblem(boundaryMask, laplacian, boundaryValue, val, col, ptr, rhs);

    std::vector<Float> x;
    x.reserve(n2);
    x.insert(x.end(), initialGuess.data(), initialGuess.data() + n2);

    Backend::params bprm {};
    hipsparseCreate(&bprm.cusparse_handle);

//    TimePoint t1 = Clock::now();
//
//    int * d_col;
//    int * d_ptr;
//    int * d_val;
//
//    hipMalloc(&d_col, sizeof(int) * col.size());
//    hipMalloc(&d_ptr, sizeof(int) * ptr.size());
//    hipMalloc(&d_val, sizeof(Float) * val.size());
//
//    hipDeviceSynchronize();
//
//    hipMemcpy(d_col, col.data(), col.size(), hipMemcpyHostToDevice);
//    hipMemcpy(d_ptr, ptr.data(), ptr.size(), hipMemcpyHostToDevice);
//    hipMemcpy(d_val, val.data(), val.size(), hipMemcpyHostToDevice);
//
//    hipDeviceSynchronize();
//
//    hipsparseSpMatDescr_t matA;
//
//    hipsparseCreateCsr(&matA, n2, n2, static_cast<int>(val.size()),
//                                      d_ptr, d_col, d_val,
//                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
//
//    hipDeviceSynchronize();
//
//    TimePoint t2 = Clock::now();
//    Float t3 = std::chrono::duration_cast<Duration>(t2 - t1).count();
//    std::cout << "cusparse initialization took " << t3 << " ms\n";

    auto f_b = Backend::copy_vector(rhs, bprm);
    auto x_b = Backend::copy_vector(x, bprm);

    Solver::params prm {};
    prm.solver.tol = relativeTolerance;

    auto t1 = Clock::now();

    Solver solve(std::tie(n2, ptr, col, val), prm, bprm);

    auto t2 = Clock::now();

    auto [iters, error] = solve(*f_b, *x_b);

    auto t3 = Clock::now();
    auto assemblyTime= std::chrono::duration_cast<Duration>(t2 - t1).count();
    auto solvingTime = std::chrono::duration_cast<Duration>(t3 - t2).count();
    auto totalTime = std::chrono::duration_cast<Duration>(t3 - t1).count();

    thrust::copy(x_b->begin(), x_b->end(), x.begin());

    return pybind11::make_tuple(
            iters,
            error,
            assemblyTime,
            solvingTime,
            totalTime,
            pybind11::array_t<Float>(n2, x.data())
    );
}

}  // namespace anonymous


void init_py_module(pybind11::module_ & m)
{
    namespace py = pybind11;
    using py::literals::operator""_a;

    // This statement must be in the same source file of the function referenced!
    // Otherwise, there will be undefined symbols.
    m.def("amgcl_solve",
          amgcl_solve,
          "boundary_mask"_a,
          "laplacian"_a,
          "boundary_value"_a,
          "initial_guess"_a,
          "relative_tolerance"_a,
          py::return_value_policy::move
    );

}

}  // namespace cpmg::cpamgcl
